#include <stdio.h>
#include <stdlib.h>
#include <set>

#include <math.h>
#include <hip/hip_runtime.h>
#include "hipblas.h"
#include <hipsolver.h>
#include <string>
#include <vector>
#include <fstream>
#include <iostream>
#include <iomanip>
#include <sstream>
#include <iterator>
#include <chrono>

struct MatrixData
{
	std::vector<std::vector<int>> edges;
	std::set<int> uniqueNodes;
};

// Function Prototypes
std::vector<std::vector<float>> eigen(std::vector<std::vector<int>>);
int doCudaStuff(std::vector<std::vector<int>>);
std::vector<std::vector<int>> createLaplacian(MatrixData);
MatrixData readEdges(std::string);
void writeCoordToFile(std::vector<std::vector<float>>, int, std::string);

// Entry point
int main(int argc, char **argv) {
	char * matrixPath = argv[1];
	int dim = std::stoi((std::string)argv[2]);

	auto start_1 = std::chrono::high_resolution_clock::now();
	MatrixData matrix_data = readEdges(matrixPath);
	auto stop_1 = std::chrono::high_resolution_clock::now();
	auto duration_1 = std::chrono::duration_cast<std::chrono::microseconds>(stop_1 - start_1);
	printf("Time to read file: %0.5f \n", (float)duration_1.count() / 1000000);

	auto start_2 = std::chrono::high_resolution_clock::now();
	std::vector<std::vector<int>> laplacian_matrix = createLaplacian(matrix_data);
	auto stop_2 = std::chrono::high_resolution_clock::now();
	auto duration_2 = std::chrono::duration_cast<std::chrono::microseconds>(stop_2 - start_2);
	printf("Time to generate matricies: %0.5f \n", (float)duration_2.count() / 1000000);

	auto start_3 = std::chrono::high_resolution_clock::now();
	std::vector<std::vector<float>> coordinates = eigen(laplacian_matrix);
	auto stop_3 = std::chrono::high_resolution_clock::now();
	auto duration_3 = std::chrono::duration_cast<std::chrono::microseconds>(stop_3 - start_3);
	printf("Time to generate Eigen: %0.5f \n", (float)duration_3.count() / 1000000);

	std::string outfilename = matrixPath;
	outfilename.append(".eigen");
	auto start_4 = std::chrono::high_resolution_clock::now();
	writeCoordToFile(coordinates, dim, outfilename);
	auto stop_4 = std::chrono::high_resolution_clock::now();
	auto duration_4 = std::chrono::duration_cast<std::chrono::microseconds>(stop_4 - start_4);
	printf("Time to write coords to File: %0.5f \n", (float)duration_4.count() / 1000000);
}

MatrixData readEdges(std::string edgePath) {
	std::vector<std::vector<int>> edges;
	std::vector<int> edgeset_1;
	std::vector<int> edgeset_2;
	std::set<int> unique_nodes;
	std::ifstream edgeFile(edgePath);
	std::string line;
	bool isLeadingEdge = true;
	if (edgeFile.is_open()) {
		while (std::getline(edgeFile, line)) {
			std::stringstream lineStream(line);
			int value;
			while (lineStream >> value) {
				unique_nodes.insert(value);
				if (isLeadingEdge) {
					edgeset_1.push_back(value);
					isLeadingEdge = false;
				}
				else {
					edgeset_2.push_back(value);
					isLeadingEdge = true;
				}
			}
		}
		edgeFile.close();
	}
	edges.push_back(edgeset_1);
	edges.push_back(edgeset_2);
	MatrixData returnData;
	returnData.edges = edges;
	returnData.uniqueNodes = unique_nodes;
	return returnData;
}

std::vector<std::vector<int>> createLaplacian(MatrixData input_matrix)
{
	// Create an empty matrix
	std::vector<int> mat_dim;
	mat_dim.push_back(input_matrix.uniqueNodes.size());

	std::vector<int> matrix(input_matrix.uniqueNodes.size()*input_matrix.uniqueNodes.size());

	std::vector<int> edgeset_1 = input_matrix.edges[0];
	std::vector<int> edgeset_2 = input_matrix.edges[1];

	// Create the laplacian edges
	for (int i = 0; i < edgeset_1.size(); ++i) {
		matrix[(edgeset_1[i] - 1) * mat_dim[0] + (edgeset_2[i] - 1)] = -1;
		matrix[(edgeset_2[i] - 1) * mat_dim[0] + (edgeset_1[i] - 1)] = -1;
	}

	// Create the laplacian degrees (diagonal)
	int degree_index = 0;
	int curSum = 0;
	for (int i = 0; i < matrix.size(); i = i + mat_dim[0]) {
		for (int k = 0; k < mat_dim[0]; k++)
		{
			if (matrix[i + k] == -1) {
				curSum++;
			}
		}
		matrix[i + degree_index] = curSum;
		degree_index++;
		curSum = 0;
	}

	std::vector<std::vector<int>> matrix_data;
	matrix_data.push_back(mat_dim);
	matrix_data.push_back(matrix);
	return matrix_data;
}

std::vector<std::vector<float>> eigen(std::vector<std::vector<int>> matrix_data)
{
	hipsolverHandle_t cusolverH;
	hipsolverStatus_t cusolver_status = HIPSOLVER_STATUS_SUCCESS;
	hipError_t cudaStat = hipSuccess;

	std::vector<int> mat_dim = matrix_data[0];
	std::vector<int> mat = matrix_data[1];

	int m = mat_dim[0];
	int lda = mat_dim[0];

	float *A; // mxm matrix
	float *V; // mxm matrix of eigenvectors
	float *W; // m- vector of eigenvalues

			  // prepare memory on the host
	A = (float *)malloc(lda*m * sizeof(float));
	V = (float *)malloc(lda*m * sizeof(float));
	W = (float *)malloc(m * sizeof(float));

	// define array to be all elements from the matrix
	for (int i = 0; i < lda*m; i++) A[i] = (float)mat[i]; //TODO is the cast safe?

														  // declare arrays on the device
	float *d_A; // mxm matrix A on the device
	float *d_W; // m- vector of eigenvalues on the device
	int *devInfo; // info on the device
	float *d_work; // workspace on the device
	int lwork = 0; // workspace size
	int info_gpu = 0; // info copied from device to host

					  // create cusolver handle
	cusolver_status = hipsolverDnCreate(&cusolverH);

	// prepare memory on the device
	cudaStat = hipMalloc((void **)& d_A, sizeof(float)* lda*m);
	cudaStat = hipMalloc((void **)& d_W, sizeof(float)*m);
	cudaStat = hipMalloc((void **)& devInfo, sizeof(int));
	cudaStat = hipMemcpy(d_A, A, sizeof(float)* lda*m, hipMemcpyHostToDevice); // copy A- >d_A 

																				 // compute eigenvalues and eigenvectors
	hipsolverEigMode_t jobz = HIPSOLVER_EIG_MODE_VECTOR;

	// use lower left triangle of the matrix
	hipblasFillMode_t uplo = HIPBLAS_FILL_MODE_LOWER;

	// compute buffer size and prepare workspace
	cusolver_status = hipsolverDnSsyevd_bufferSize(cusolverH, jobz, uplo, m, d_A, lda, d_W, &lwork);
	cudaStat = hipMalloc((void **)& d_work, sizeof(float)* lwork);

	// compute the eigenvalues and eigenvectors for a symmetric ,
	// real mxm matrix ( only the lower left triangle af A is used )
	cusolver_status = hipsolverDnSsyevd(cusolverH, jobz, uplo, m,
		d_A, lda, d_W, d_work, lwork, devInfo);
	cudaStat = hipDeviceSynchronize();

	cudaStat = hipMemcpy(W, d_W, sizeof(float)*m, hipMemcpyDeviceToHost); // copy d_W - >W
	cudaStat = hipMemcpy(V, d_A, sizeof(float)* lda*m, hipMemcpyDeviceToHost); // copy d_A - >V
	cudaStat = hipMemcpy(&info_gpu, devInfo, sizeof(int), hipMemcpyDeviceToHost); // copy devInfo - > info_gpu

	int counter = 0;
	float curvec = V[counter];
	std::vector<std::vector<float>> coordinates;
	std::vector<float> curDim;
	while (counter != m * m) {
		curDim.push_back(V[counter]);
		++counter;
		if (counter % m == 0) {
			coordinates.push_back(curDim);
			curDim.clear();
		}
	}

	// free memory
	hipFree(d_A);
	hipFree(d_W);
	hipFree(devInfo);
	hipFree(d_work);
	hipsolverDnDestroy(cusolverH);
	hipDeviceReset();
	if (cusolver_status == 0)
	{
		return coordinates;
	}
	else {
		printf("Error computing Eigen values: \n");
		printf("CuSolver returned %i ", cusolver_status);
		exit(1);
	}
}

void writeCoordToFile(std::vector<std::vector<float>> coordinates, int dim, std::string outfilename) {
	std::ofstream out_file;
	out_file.open(outfilename);
	
	for (int i = 1; i <= dim; ++i) {
		for (int k = 0; k < coordinates[i].size(); k++) {
			out_file << std::fixed << std::setprecision(8) << coordinates[i][k] << " ";
			//printf("%0.12f ", coordinates[i][k]);
		}
		out_file << "\n";
		//printf("\n");
	}
	out_file.close();
}